#include "hip/hip_runtime.h"
/**
 * @author NageshAC
 * @email nagesh.ac.aralaguppe@fau.de
 * @create date 2021-07-28 09:49:27
 * @modify date 2021-07-28 09:49:27
 * @desc contains definitions of all smoothening kernels.
 */


#pragma once

#include<cmath>
#include<hip/hip_runtime.h>

#include "./includes/operators.cu"

#define M_PI 3.14159265358979323846

using namespace thrust;

//**************************************************
// W_poly6, grad_poly6 and lap_poly6
//**************************************************

__device__
inline double w_poly6 (const double* r, const double POLY6, const double h){
    double n_r2 = norm2(r);
    double h2 = pow(h,2);
    double result;
    if(n_r2<=h2){
        result = pow(h2 - n_r2, 3);
    }
    else return 0;
    result *= POLY6;
    return result;
}

__device__
inline void grad_poly6(double* r, const double GPOLY6, const double h){
    double n_r2 = norm2(r);
    double h2 = pow(h,2);
    double result = 0;
    if(n_r2<=h2){
        result = pow(h2 - n_r2, 2);
        result *= GPOLY6;
    }
    multiply(&result, r);
}

__device__
inline double lap_poly6(const double* r, const double GPOLY6, const double h){
    double n_r2 = norm2(r);
    double h2 = pow(h,2);
    if(n_r2<=h2){
        double result;
        result = h2 - n_r2;
        result *= (3*h2-7*n_r2);
        return GPOLY6*result;
    }
    return 0;
}

//**************************************************
//  grad_spiky  for pressure field
//**************************************************
__device__
inline void grad_spiky(double* r, const double PV, const double h){
    double n_r = norm(r);
    if(0<=n_r && n_r<=h){
        double result = pow(h-n_r, 2) / (n_r*1000);
        result *= -PV;
        multiply(&result, r);
    }
    else{
        for(int i=0; i<3; i++) r[i] = 0;
    }
}

//**************************************************
//  lap_viscosity  for viscosity field
//**************************************************
__device__
inline double lap_viscosity(const double* r, const double PV, const double h){
    double n_r = norm(r);
    if(0<=n_r && n_r<=h){
        double result = h-n_r;
        result *= PV;
        return result;
    }
    return 0;
}
