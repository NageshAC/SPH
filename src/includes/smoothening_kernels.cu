#include "hip/hip_runtime.h"
/**
 * @author NageshAC
 * @email nagesh.ac.aralaguppe@fau.de
 * @create date 2021-07-28 09:49:27
 * @modify date 2021-07-28 09:49:27
 * @desc contains definitions of all smoothening kernels.
 */


#pragma once

#include<cmath>
#include<thrust/device_vector.h>
#include<hip/hip_runtime.h>

#include "./includes/particle.cpp"
#include "./includes/operators.cu"

#define M_PI 3.14159265358979323846

using namespace thrust;

//**************************************************
// W_poly6, grad_poly6 and lap_poly6
//**************************************************
__device__
inline double w_poly6 (const double* r, const double h){
    double n_r = norm(r);
    double result;
    if(0<=n_r && n_r<=h){
        result = pow(pow(h,2) - pow(n_r,2), 3);
    }
    else if(n_r > h){
        return 0;
    }
    double c = 315/(64*M_PI*pow(h,9));
    result *= c;
    return result;
    
}

__device__
inline void grad_poly6(double* r, const double h){
    double n_r = norm(r);
    if(0<=n_r && n_r<=h){
        double c = -945/(32*M_PI*pow(h,9));
        double result;
        result = pow(pow(h,2) - pow(n_r,2), 2);
        result *= c;
        multiply(&result, r);
    }
}

__device__
inline double lap_poly6(const double* r, const double h){
    double n_r = norm(r);
    if(0<=n_r && n_r<=h){
        double c = -945/(32*M_PI*pow(h,9));
        double result;
        result = pow(h,2) - pow(n_r,2);
        result *= 3*pow(h,2)-7*pow(n_r,2);
        return c*result;
    }
    return 0;
}

//**************************************************
//  grad_spiky  for pressure field
//**************************************************
__device__
inline void grad_spiky(double* r, const double h){
    double n_r = norm(r);
    if(0<n_r && n_r<=h){
        double c = -45/(M_PI*pow(h,6)*n_r);
        double result = pow(h-n_r, 2);
        result *= c;
        multiply(&result, r);
    }
    else{
        for(int i=0; i<3; i++) r[i] = 0;
    }
}

//**************************************************
//  lap_viscosity  for viscosity field
//**************************************************
__device__
inline double lap_viscosity(const double* r, const double h){
    double n_r = norm(r);
    if(0.05<=n_r && n_r<h){
        double c = 45/(M_PI*pow(h,6));
        double result = h-n_r;
        result *= c;
        return result;
    }
    return 0;
}