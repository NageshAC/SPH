/**
 * @author NageshAC
 * @email nagesh.ac.aralaguppe@fau.de
 * @create date 2021-08-10 12:11:29
 * @modify date 2021-08-10 12:11:29
 * @desc Contains few operators definitions used in this project.
*/

#pragma once
#include<iostream>
#include<cmath>

#include<hip/hip_runtime.h>
//**************************************************************
// copy function
//**************************************************************
__device__ __host__
inline void copy(double* x, const double* y, int n = 3){
    for(int i=0; i<n; i++)
        x[i] = y[i];
}

//**************************************************************
// l2 norm
//**************************************************************
__device__
inline double norm(const double* x, int dim = 3){
    double result = 0;
    for(int i=0; i<dim; i++){
        result += pow(x[i],2);
    }
    return sqrt(result);
}
__device__
inline double norm2(const double* x, int dim = 3){
    double result = 0;
    for(int i=0; i<dim; i++){
        result += pow(x[i],2);
    }
    return (result);
}

//**************************************************************
// vector const multiplication
//**************************************************************
__device__
inline void axpy(const double a, const double* x, double* y, int dim =3){
    for(int i=0; i<3; i++) y[i] += a * x[i];

}

//**************************************************************
// vector const multiplication
//**************************************************************
__device__
inline void multiply(const double* c, double* x, int dim = 3){
    for(auto i=0; i<dim; i++) x[i] *= *c;
}

__device__
inline void multiply(double* r, const double c, const double* x, int dim = 3){
    for(auto i=0; i<dim; i++) r[i] = x[i] * (c);
}

//**************************************************************
// vector vector subtraction
//**************************************************************
__device__
inline void subtract(double* x, const double* y, int dim = 3){
    for(int i=0; i<3; i++) x[i] = x[i] - y[i];
}

__device__
inline void subtract(double* r, const double* x, const double* y, int dim = 3){
    for(int i=0; i<3; i++) r[i] = x[i] - y[i];
}

//**************************************************************
// vector vector addition
//**************************************************************
__device__
inline void add(double* x, const double* y, int dim = 3){
    for(int i=0; i<3; i++) x[i] = x[i] + y[i];
}

__device__
inline void add(double* r, const double* x, const double* y, int dim = 3){
    for(int i=0; i<3; i++) r[i] = x[i] + y[i];
}