#include "hip/hip_runtime.h"
/**
 * @author NageshAC
 * @email nagesh.ac.aralaguppe@fau.de
 * @create date 2021-08-10 12:19:30
 * @modify date 2021-08-10 12:19:30
 * @desc Contains definitions of field calculation such as force, Pressure ...
 */

 #pragma once

#include<hip/hip_runtime.h>
#include"./particle.cpp"
#include"./smoothening_kernels.cu"
#include"./operators.cu"    
#include"./check.cu"

//************************************************************
// density calculation
//************************************************************
__global__
void cal_density(particle* p, double ro_0, int N, double h){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<N){
        double den = 0;
        for(int j=0; j<N; j++){
            double r[3]; 
            subtract(r,p[idx].g_position(),p[j].g_position());
            den += p[j].g_mass() * w_poly6(r, h);
        }
        p[idx].s_density(den);
        p[idx].update_md();
    }
}

//************************************************************
// force calculation
//************************************************************
__global__
void cal_force(
    particle* p, const double* g, 
    double ro_0, double k, double mu,
    double sigma, double l, int N, double h
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<N){

        double *fi = p[idx].g_force();
        double *xi = p[idx].g_position();
        double roi = p[idx].g_density();

        // p[idx].reset_force();

        // gravitational force
        p[idx].s_force(g);  // reset force
        multiply(&ro_0, fi);
        __syncthreads();

        // pressure
        {
            p[idx].s_pressure(k*(p[idx].g_density()-ro_0));
            __syncthreads();
            double pr[3], r[3], c;
            for(int i=0; i<3; i++) pr[i] = 0;
            for(int j=0; j<N; j++){
                if(idx != j){
                    subtract(r,xi,p[j].g_position());
                    if(norm(r) <= h){
                        c = -1*(p[idx].g_pressure() + p[j].g_pressure())*p[j].g_md()/2;
                        grad_spiky(r, h);
                        axpy(c, r, pr);
                    }
                }
            }
            // printf("P(%d) = %lf %lf %lf\n", idx, pr[0], pr[1], pr[2]);
            add(fi, pr);
        }
        __syncthreads();

        // viscosity
        {
            double vs[3], del_v[3], r[3], c;
            for(int i=0; i<3; i++) vs[i] = 0;
            for(int j=0; j<N; j++){
                if(idx != j){
                    subtract(r,xi,p[j].g_position());
                    subtract(del_v, p[j].g_velocity(), p[idx].g_velocity());
                    // if(idx==0 && j==1) printf("lap = %lf md = %lf\n", lap_viscosity(r, h),p[j].g_md());
                    c = p[j].g_md() * lap_viscosity(r, h);
                    axpy(c, del_v, vs);
                    // if(idx==0 && j==1) printf("c = %lf vs = %lf %lf  %lf\n", c, vs[0], vs[1], vs[2]);
                }
            }
            axpy(mu, vs, fi);
        }
        __syncthreads();


        // surface tension
        {
            // calculating c(i)
            double color = 0;
            double r[3];
            for(int j=0; j<N; j++){
                // if(idx != j){ 
                    subtract(r,p[idx].g_position(),p[j].g_position());
                    color += p[j].g_md() * lap_poly6(r, h);
                // }
            }
            p[idx].s_color(color);

            // calculating n(i)
            double n[3];
            for(int i=0; i<3; i++) n[i] = 0;
            for(int j=0; j<N; j++){
                subtract(r,p[idx].g_position(),p[j].g_position());
                grad_poly6(r,h);
                axpy(p[j].g_md(), r, n);
            }
            p[idx].s_n(n);

            // calculating surdace force
            double n_n = norm(p[idx].g_n());
            if(n_n >= l){
                color *= -sigma/n_n;
                axpy(color, n, fi);
            }
        }
        __syncthreads();
    }
}

//************************************************************
// leap-frog scheme of integration
//************************************************************
__global__
void cal_leapfrog(
    particle* p, 
    const double xmin, const double xmax,
    const double ymin, const double ymax, 
    const double zmin, const double zmax,
    const double del_t, const int N){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<N){
        double den = p[idx].g_density();
        double *fi = p[idx].g_force();
        if(den>0){

            double c = del_t/(2*den);

            axpy(c, fi, p[idx].g_velocity());

            axpy(del_t, p[idx].g_velocity(), p[idx].g_position());

            // boundary condition
            check_bound(
                p[idx].g_position(), p[idx].g_velocity(), del_t,
                xmin, xmax, ymin, ymax, zmin, zmax
            );

            axpy(c, fi, p[idx].g_velocity());

        }
    }
}