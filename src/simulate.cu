/**
 * @author NageshAC
 * @email nagesh.ac.aralaguppe@fau.de
 * @create date 2021-08-09 21:21:59
 * @modify date 2021-08-09 21:21:59
 * @desc main file: contains all the function calls and kernel calls
 */

 #include<iostream>
 #include<string>

 #include<hip/hip_runtime.h>
 #include<thrust/host_vector.h>
 #include<thrust/device_vector.h>

 #include "./includes/parameters.cpp"
 #include "./includes/particle.cpp"
 #include "./includes/input.cpp"
 #include "./includes/wireframe.cpp"

 using namespace std;
 using namespace thrust;

 int main(){

    string input_path = "./data/";
    string output_path = "./sln/";
    string par_file = "water.par";

    string full_file = input_path + par_file;

    string part_in_file, vtk_out_file;
    int vtk_out_freq;
    double time_end, del_t, ro_0, mu, sigma, l, 
        k, h, x_min, x_max, y_min, y_max, z_min, z_max;
    host_vector<double> g(3,0);

    //**************************************************
    // reading .par file
    //**************************************************
    {
        readParam(
            full_file,
            part_in_file, vtk_out_freq,
            vtk_out_file, time_end, del_t, 
            raw_pointer_cast(&g[0]), 
            ro_0, mu,
            sigma, l, k, h,
            x_min, x_max,
            y_min, y_max,
            z_min, z_max
        );
        // printParam(
        //     part_in_file, vtk_out_freq,
        //     vtk_out_file, time_end, del_t, 
        //     raw_pointer_cast(&g[0]), 
        //     ro_0, mu,
        //     sigma, l, k, h,
        //     x_min, x_max,
        //     y_min, y_max,
        //     z_min, z_max
        // );
    }

    host_vector<particle> p;
    int N, frames = (time_end/del_t);

    //**************************************************
    // reading .in files
    //**************************************************
    {
        full_file = input_path + part_in_file;
        p = readInput(full_file,N);
        // printInput(
        //     raw_pointer_cast(&p[0]),
        //     N
        // );
    }

    //**************************************************
    // boundary 
    //**************************************************
    create_wireframe(
        x_min, x_max, 
        y_min, y_max, 
        z_min, z_max
    );

    return 0;
 }